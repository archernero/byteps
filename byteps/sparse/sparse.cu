// Copyright 2020 Bytedance Inc. or its affiliates. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// =============================================================================

#define BYTEPS_DEBUG

#include "sparse.h"
#include "sparse_dense.h"
#include "sparse.cuh"
#include <iostream>

namespace byteps {
namespace sparse {

/**
  embedBuffers: the addresses of all embedding buffers (could have variable length)
  denseBuffers: the addresses of all dense buffers (the length should be identical)
  embedBufferLens: the length of the embedding buffers (could have variable length)
  size: the length of a dense buffer (in bytes), it is equivalent for all GPUs
 */
void bytepsSparseInit(std::vector<void*>& embedBuffers, 
                      std::vector<void*>& denseBuffers, 
                      std::vector<size_t>& embedBufferLens, 
                      size_t denseBufferLen) {
  BytePSSparseCommon::Init();
  CHECK_EQ(embedBuffers.size(), denseBuffers.size());
  CHECK_EQ(embedBufferLens.size(), denseBuffers.size());

  // Init IPC stuff
  volatile shmStruct *shm = NULL;
  sharedMemoryInfo info;
  CHECK_EQ(sharedMemoryCreate(bpsShmName, sizeof(*shm), &info), 0);
  shm = (volatile shmStruct *)info.addr;
  memset((void *)shm, 0, sizeof(*shm));

  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  auto globalSize = localSize * workerNum;

  for (int i = 0; i < localSize; i++) {
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, i));

    // CUDA IPC is only supported on devices with unified addressing
    CHECK(prop.unifiedAddressing)
        << "Device " << i << " does not support unified addressing.";

    shm->devices[shm->nprocesses++] = i;
    CHECK_GT(MAX_CUDA_DEVICES, shm->nprocesses);
  }
  CHECK(shm->nprocesses > 0) 
      << "No cuda device suppported";
  CHECK_EQ(shm->nprocesses, embedBuffers.size())
      << "Shared memory processes: " << shm->nprocesses 
      << ", send buffers: " << embedBuffers.size();

  _embedBuffers.assign(embedBuffers.begin(), embedBuffers.end());
  _denseBuffers.assign(denseBuffers.begin(), denseBuffers.end());

  _localEmbedBufLens.resize(localSize);
  _globalEmbedBufLens.resize(workerNum, std::vector<size_t>(localSize));
  _globalTotalEmbedBufLens.resize(workerNum, 0);

  // Allocate memory for each process and fill 
  // the shared memory buffer with the IPC handles 
  for (size_t i = 0; i < shm->nprocesses; i++) {
    CUDA_CALL(hipSetDevice(
        shm->devices[i]));
    CUDA_CALL(hipIpcGetMemHandle(
        (hipIpcMemHandle_t *)&shm->embedMemHandle[i], embedBuffers[i]));
    
    shm->embedBufferLength[i] = embedBufferLens[i];
    // Store the buffers 
    _localEmbedBufLens[i] = embedBufferLens[i]; // local buffer length
  }
  _denseBufferLen = denseBufferLen;
  shm->denseBufferLength = denseBufferLen;

#ifdef BYTEPS_DEBUG
  // For debug: print _localEmbedBufLens
  std::cout << "_localEmbedBufLens:" << std::endl;
  for (auto len : _localEmbedBufLens) 
    std::cout << len << " ";
  std::cout << std::endl;
#endif

  for (int i = 0; i < localSize; i++) {
    _globalEmbedBufLens[workerID][i] = _localEmbedBufLens[i];
  }
  
  // The followings are for the global coordination of 
  // the embedding buffer length, which is equivalent to all-gather 
  auto ps = BytePSSparseCommon::GetPS();
  if (BytePSSparseCommon::IsDistributed()) {
    CHECK(ps); // must init the pslite instance before
    
    // keys
    std::vector<ps::Key> pskeys(workerNum);
    std::vector<ps::SArray<ps::Key>> keys_array; 

    // lens
    std::vector<int> pslens(workerNum);
    std::vector<ps::SArray<int>> lens_array; 

    // vals
    std::vector<ps::SArray<char>> vals_array; 

    auto krs = ps::Postoffice::Get()->GetServerKeyRanges();
    for (int i = 0; i < workerNum; i++) {
      ps::Key key = i;
      int server = i;
      
      // keys 
      pskeys[i] = krs[server].begin() + key;
      ps::SArray<ps::Key> keys;
      keys.reset(&pskeys[i], 1, [](void *){});
      keys_array.push_back(keys);
      
      // lens 
      pslens[i] = sizeof(size_t) * localSize;
      ps::SArray<int> lens;
      lens.reset(&pslens[i], 1, [](void *){});
      lens_array.push_back(lens);

      // vals 
      ps::SArray<char> vals;
      vals.reset((char*)_globalEmbedBufLens[i].data(), localSize * sizeof(size_t), [](void *){});
      vals_array.push_back(vals);
    }

    // Push once to the associated server
    {
      int server = workerID;
      auto keys = keys_array[server];
      auto vals = vals_array[server];
      auto lens = lens_array[server];
      ps->Wait(ps->ZPush(keys, vals, lens));
    }

    ps::Postoffice::Get()->Barrier(
        0, ps::kWorkerGroup + ps::kServerGroup + ps::kScheduler);

    // Pull the embedding buffer length of other workers
    for (int i = 0; i < workerNum; i++) {
      if (i == workerID) continue; // skip myself
      int server = i;
      auto keys = keys_array[server];
      auto vals = vals_array[server];
      auto lens = lens_array[server];
      ps->Wait(ps->ZPull(keys, &vals, &lens));
    }
  } // BytePSSparseCommon::IsDistributed()

  for (int wid = 0; wid < workerNum; wid++) {
    for (int gpu = 0; gpu < localSize; gpu++) {
      _globalTotalEmbedBufLens[wid] += _globalEmbedBufLens[wid][gpu];
    }
  }

#ifdef BYTEPS_DEBUG
  // For debug: print _globalEmbedBufLens
  std::cout << "_globalEmbedBufLens:" << std::endl;
  for (auto vec : _globalEmbedBufLens) {
    for (auto len : vec) {
      std::cout << len << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;

  // For debug: print _globalTotalEmbedBufLens
  std::cout << "_globalTotalEmbedBufLens:" << std::endl;
  for (auto len : _globalTotalEmbedBufLens) {
    std::cout << len << " ";
  }
  std::cout << std::endl;
#endif 

  // Check the buffer size 
  size_t accmul = 0;
  for (auto len : _globalTotalEmbedBufLens) accmul += len / globalSize;
  CHECK_EQ(accmul, _denseBufferLen) << accmul << " " << _denseBufferLen;

  // Calc the global offset for the communication buffers
  size_t global_offset = 0;
  for (int id = 0; id < workerID; id++) {
    global_offset += _globalTotalEmbedBufLens[id] / globalSize;
  }

  // Prepare gossip-gather communication
  _local_gather_comms.resize(localSize);
  for (int i = 0; i < localSize; i++) {
    std::vector<float*> srcs(localSize);
    std::vector<size_t> srcs_lens(localSize);
    std::vector<size_t> send_counts(localSize);

    for (int j = 0; j < localSize; j++) {
      srcs[j] = 
          (float*)_embedBuffers[j] + 
          _localEmbedBufLens[j] / globalSize * (i + localSize * workerID);

      srcs_lens[j] = 
          _localEmbedBufLens[j] / globalSize * 
          (globalSize - (i + localSize * workerID));
          
      send_counts[j] = 
          _localEmbedBufLens[j] / globalSize;
    }
    float* dst = (float *)_denseBuffers[i] + global_offset;
    size_t dst_len = _globalTotalEmbedBufLens[workerID] / globalSize;

    std::string planfile_name("gather_plan_");
    planfile_name += std::to_string(i) + std::string(".json");
    _local_gather_comms[i] = std::make_unique<LocalGatherComm>(
        planfile_name, localSize, srcs, srcs_lens, send_counts, dst, dst_len);
  }

  // Prepare gossip-scatter communication
  _local_scatter_comms.resize(localSize);
  for (int i = 0; i < localSize; i++) {
    float* src = (float *)_denseBuffers[i] + global_offset;
    size_t src_len = _globalTotalEmbedBufLens[workerID] / globalSize;
    std::vector<float*> dsts(localSize);
    std::vector<size_t> dsts_lens(localSize);
    std::vector<size_t> send_counts(localSize);
    for (int j = 0; j < localSize; j++) {
      dsts[j] = 
          (float*)_embedBuffers[j] + 
          _localEmbedBufLens[j] / globalSize * (i + localSize * workerID);

      dsts_lens[j] = 
          _localEmbedBufLens[j] / globalSize * 
          (globalSize - (i + localSize * workerID));

      send_counts[j] = 
          _localEmbedBufLens[j] / globalSize;
    }

    std::string planfile_name("scatter_plan_");
    planfile_name += std::to_string(i) + std::string(".json");
    _local_scatter_comms[i] = std::make_unique<LocalScatterComm>(
        planfile_name, localSize, src, src_len, send_counts, dsts, dsts_lens);
  }
  
  if (BytePSSparseCommon::IsDistributed()) {
    // Prepare distributed gather communication
    _dist_gather_comms.resize(localSize);
    for (int i = 0; i < localSize; i++) {
      auto ps = BytePSSparseCommon::GetPS();
      _dist_gather_comms[i] = std::make_unique<DistGatherComm>(ps, _globalEmbedBufLens, 
        _denseBuffers[i], _denseBufferLen, i, localSize, workerID, workerNum);
    }
    // Prepare distributed scatter communication
    _dist_scatter_comms.resize(localSize);
    for (int i = 0; i < localSize; i++) {
      auto ps = BytePSSparseCommon::GetPS();
      _dist_scatter_comms[i] = std::make_unique<DistScatterComm>(ps, _globalEmbedBufLens, 
        _denseBuffers[i], _denseBufferLen, i, localSize, workerID, workerNum);
    }
  } 
}

/*
void bytepsSparseInitDense(std::vector<void*>& denseDeltaBeforeReduceBuffers,
                           std::vector<void*>& denseDeltaAfterReduceBuffers,
                           int sizeDenseDelta) {

  CHECK_EQ(denseDeltaBeforeReduceBuffers.size(), denseDeltaAfterReduceBuffers.size());
  _denseDeltaBufferLength = sizeDenseDelta;

  // // Init IPC stuff
  // sharedMemoryInfo info;
  // CHECK_EQ(sharedMemoryCreate(bpsShmName, sizeof(shmStruct), &info), 0);
  // auto shm = (volatile shmStruct *)info.addr;
  // memset((void *)shm, 0, sizeof(*shm));

  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();

  // for (int i = 0; i < localSize; i++) {
  //   hipDeviceProp_t prop;
  //   CUDA_CALL(hipGetDeviceProperties(&prop, i));

  //   // CUDA IPC is only supported on devices with unified addressing
  //   if (!prop.unifiedAddressing) {
  //     // BPS_LOG(INFO) << "Device " << i << " does not support unified addressing, skipping...";
  //     continue;
  //   }
  //   // We require two processes accessing each device, so we need
  //   // to ensure exclusive or prohibited mode is not set
  //   if (prop.computeMode != hipComputeModeDefault) {
  //     // BPS_LOG(INFO) << "Device " << i << "is in an unsupported compute mode for this sample";
  //     continue;
  //   }

  //   shm->devices[shm->nprocesses++] = i;
  //   CHECK_GT(MAX_CUDA_DEVICES, shm->nprocesses);
  // }

  // CHECK(shm->nprocesses > 0) 
  //     << "No cuda device suppported";
  // CHECK_EQ(shm->nprocesses, embedBuffers.size())
  //     << "Shared memory processes: " << shm->nprocesses 
  //     << ", send buffers: " << embedBuffers.size();

  // We need to manually we need to clear the containers because
  // bytepsSparseInit() might be (unexpectedly) invoked multiple times
  _denseDeltaBeforeReduceBuffers.clear();
  _denseDeltaAfterReduceBuffers.clear();
  for (size_t i = 0; i < localSize; i++) {
    _denseDeltaBeforeReduceBuffers.push_back(denseDeltaBeforeReduceBuffers[i]); 
    _denseDeltaAfterReduceBuffers.push_back(denseDeltaAfterReduceBuffers[i]);
  }

  // Allocate memory and an event for each process and fill 
  // the shared memory buffer with the IPC handles 
  // for (size_t i = 0; i < shm->nprocesses; i++) {
  //   hipEvent_t event;
  //   CUDA_CALL(hipSetDevice(
  //       shm->devices[i]));

  //   CUDA_CALL(hipIpcGetMemHandle(
  //     (hipIpcMemHandle_t *)&shm->denseDeltaBeforeReduceMemHandle[i], denseDeltaBeforeReduceBuffers[i]));
  //   CUDA_CALL(hipIpcGetMemHandle(
  //     (hipIpcMemHandle_t *)&shm->denseDeltaAfterReduceMemHandle[i], denseDeltaAfterReduceBuffers[i]));

  //   // Store the buffers 
  //   _denseDeltaBeforeReduceBuffers.push_back(denseDeltaBeforeReduceBuffers[i]); 
  //   _denseDeltaAfterReduceBuffers.push_back(denseDeltaAfterReduceBuffers[i]);
  // }

  // Get CPU buffer for dense layer reduceasync
  {
    CUDA_CALL(hipHostAlloc(
        &_cpuDenseDeltaBuffers, sizeDenseDelta, hipHostMallocMapped | hipHostMallocPortable));
  }

  // Start the DenseReduce loop
  runDenseReduceLoop(_denseReduceLoop);
  _denseReducer = new ::byteps::common::CpuReducer(nullptr);
}
*/

extern "C" void bytepsSparseInitDensePerGPU(int device_id /* starts with 0 */,
                                            void* denseDeltaBeforeReduceBuffer,
                                            void* denseDeltaAfterReduceBuffer,
                                            int sizeDenseDelta) {
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  assert((device_id < localSize) && "Device id must be within local gpu size.");

  std::cout << "Init BytePS Sparse for dense layers: Device" << device_id << std::endl;

  if (device_id == 0){
    _denseDeltaBufferLength = sizeDenseDelta;
    _mtx_DenseLatestBuffers = new std::mutex();

    // Allocate latest parameter buffer.
    CUDA_CALL(hipHostAlloc(
      &_cpuDenseLatestBuffers, sizeDenseDelta, hipHostMallocMapped | hipHostMallocPortable));

    // Start the DenseReduce loop
    // _denseReducer = new ::byteps::common::CpuReducer(nullptr);
    // runDenseReduceLoop(_denseReduceLoop);

    // Start the 3-stage pipeline: D2H -> CpuReduce -> H2D
    _denseReducer = new ::byteps::common::CpuReducer(nullptr);
    runDenseReducePipeline(_denseD2HLoop, _denseReduceLoop, _denseH2DLoop, _denseReducer, _mtx_DenseLatestBuffers);
  } else{
    CHECK_EQ(_denseDeltaBufferLength, sizeDenseDelta);
  }

  // Get CPU buffer for dense layer reduceasync
  void * _cpuDenseDeltaBuffers_per_gpu;
  CUDA_CALL(hipHostAlloc(
    &_cpuDenseDeltaBuffers_per_gpu, sizeDenseDelta, hipHostMallocMapped | hipHostMallocPortable));
  _cpuDenseDeltaBuffers.push_back(_cpuDenseDeltaBuffers_per_gpu);
  _denseDeltaBeforeReduceBuffers.push_back(denseDeltaBeforeReduceBuffer); 
  _denseDeltaAfterReduceBuffers.push_back(denseDeltaAfterReduceBuffer);

  bool is_ready = false;
  std::mutex * mtx = new std::mutex();
  std::condition_variable * signal_cv = new std::condition_variable();
  _is_ready_per_gpu.push_back(is_ready);
  _signal_mtx_per_gpu.push_back(mtx);
  _signal_cv_per_gpu.push_back(signal_cv);
}

void bytepsSparseShutdown() {
}


void bytepsGatherExecAsync(int local_rank, hipStream_t stream) {
  // Gather from local peer GPUs on the same worker
  _local_gather_comms[local_rank]->ExecAsync();
  
  // Gather from distributed peer GPUs on other workers
  if (BytePSSparseCommon::IsDistributed()) {
    _dist_gather_comms[local_rank]->ExecAsync();
  }
}

void bytepsSynchronize(int local_rank, hipStream_t stream, OP op) { 
  switch (op) {
    case GATHER: {
      _local_gather_comms[local_rank]->Sync();
      if (BytePSSparseCommon::IsDistributed()) {
        _dist_gather_comms[local_rank]->Sync();
      }
    } break;
    case SCATTER: {
      _local_scatter_comms[local_rank]->Sync();
      if (BytePSSparseCommon::IsDistributed()) {
        _dist_scatter_comms[local_rank]->Sync();
      }
    } break;
    default:
      CHECK(0) << "unrecognized operation: " << op;
  }
  CUDA_CALL(hipStreamSynchronize(stream));
}

void bytepsScatterExecAsync(int local_rank, hipStream_t stream) {
  // Scatter to local peer GPUs on the same worker
  _local_scatter_comms[local_rank]->ExecAsync();
  
  // Scatter to distributed peer GPUs on other workers
  if (BytePSSparseCommon::IsDistributed()) {
    _dist_scatter_comms[local_rank]->ExecAsync();
  }
}

// void dense_ready_callback(int local_rank) {
//   // std::mutex signal_mtx = _signal_mtx_per_gpu.at(local_rank);
//   // std::condition_variable signal_cv = _signal_cv_per_gpu.at(local_rank);

//   std::unique_lock<std::mutex> lck(* _signal_mtx_per_gpu.at(local_rank));
//   _is_ready_per_gpu.at(local_rank) = true;
//   _signal_cv_per_gpu.at(local_rank)->notify_one();
// }

// TODO (chengyu.dai): Add Broadcast for initializing the latestBuffer.
void bytepsDenseReduceExecAsync(int local_rank, hipStream_t stream) {
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  void* baseSrcPtr = (void*) (_denseDeltaBeforeReduceBuffers.at(local_rank));
  void* baseResultPtr = (void*) (_denseDeltaAfterReduceBuffers.at(local_rank));

  size_t buffer_size = _denseDeltaBufferLength;

  // Create a local thread and related mutex to synchronnize.
  _is_ready_per_gpu.at(local_rank) = false;

  // auto reduce_async_job = [//& signal_mtx, & signal_cv, & is_ready, 
  //                          local_rank, baseSrcPtr, baseResultPtr,
  //                          buffer_size, stream]() {
  //   // Copy dense layer's param delta D2H.
  //   CUDA_CALL(hipMemcpyAsync((void *)_cpuDenseDeltaBuffers, baseSrcPtr, buffer_size, hipMemcpyDeviceToHost, stream));
  //   CUDA_CALL(hipStreamSynchronize(stream));

  //   // CPU Work to reduce.
  //   _denseReducer->sum(_cpuDenseLatestBuffers, _cpuDenseDeltaBuffers, _denseDeltaBufferLength /* in bytes*/, DataType::BYTEPS_FLOAT32);

  //   // Copy dense layer's latest param H2D.
  //   CUDA_CALL(hipMemcpyAsync(baseResultPtr, _cpuDenseLatestBuffers, buffer_size, hipMemcpyHostToDevice, stream));
  //   CUDA_CALL(hipStreamSynchronize(stream));

  //   dense_ready_callback(local_rank);
  // };
  // _denseReduceLoop->add_worker(reduce_async_job);

  auto dense_ready_callback = 
    [] (int local_rank) {
    std::unique_lock<std::mutex> lck(* _signal_mtx_per_gpu.at(local_rank));
    _is_ready_per_gpu.at(local_rank) = true;
    _signal_cv_per_gpu.at(local_rank)->notify_one();
  };

  DenseTask task;
  {
    task.workerID = workerID;
    task.local_rank = local_rank;
    task.buffer_size = buffer_size; // In bytes.
    task.streamH2D = stream;
    task.streamD2H = stream; // TODO(chengyu.dai): separate the streams for two directions.

    task.baseSrcPtr = baseSrcPtr;
    task.cpuDenseDeltaPtr = (void *) (_cpuDenseDeltaBuffers.at(local_rank));
    task.cpuDenseLatestPtr = _cpuDenseLatestBuffers;
    task.baseResultPtr = baseResultPtr;

    task.allFinishCallback = dense_ready_callback;
  }
  _denseD2HLoop->add_predefined_worker(task);
}

void bytepsDenseSynchronize(int local_rank, hipStream_t stream) {
  // auto signal_mtx = _signal_mtx_per_gpu.at(local_rank);
  // std::condition_variable signal_cv = _signal_cv_per_gpu.at(local_rank);

  std::unique_lock<std::mutex> lck(* _signal_mtx_per_gpu.at(local_rank));
  while (!_is_ready_per_gpu.at(local_rank))
    _signal_cv_per_gpu.at(local_rank)->wait(lck);
}


} // namespace sparse
} // namespace byteps 