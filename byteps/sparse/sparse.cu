// Copyright 2020 Bytedance Inc. or its affiliates. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// =============================================================================

#include "sparse.h"
#include "sparse_dense.h"
#include "sparse.cuh"
#include <iostream>

namespace byteps {
namespace sparse {

/**
  embedBuffers: the addresses of all embedding buffers (could have variable length)
  denseBuffers: the addresses of all dense buffers (the length should be identical)
  embedBufferLens: the length of the embedding buffers (could have variable length)
  size: the length of a dense buffer (in bytes), it is equivalent for all GPUs
 */
void bytepsSparseInit(std::vector<void*>& embedBuffers, 
                      std::vector<void*>& denseBuffers, 
                      std::vector<size_t>& embedBufferLens, 
                      size_t denseBufferLen) {
  BytePSSparseCommon::Init();
  CHECK_EQ(embedBuffers.size(), denseBuffers.size());
  CHECK_EQ(embedBufferLens.size(), denseBuffers.size());

  // Init IPC stuff
  sharedMemoryInfo info;
  CHECK_EQ(sharedMemoryCreate(bpsShmName, sizeof(shmStruct), &info), 0);
  auto shm = (volatile shmStruct *)info.addr;
  memset((void *)shm, 0, sizeof(*shm));

  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  auto globalSize = localSize * workerNum;

  for (int i = 0; i < localSize; i++) {
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, i));

    // CUDA IPC is only supported on devices with unified addressing
    if (!prop.unifiedAddressing) {
      // BPS_LOG(INFO) << "Device " << i << " does not support unified addressing, skipping...";
      continue;
    }
    // We require two processes accessing each device, so we need
    // to ensure exclusive or prohibited mode is not set
    if (prop.computeMode != hipComputeModeDefault) {
      // BPS_LOG(INFO) << "Device " << i << "is in an unsupported compute mode for this sample";
      continue;
    }

    shm->devices[shm->nprocesses++] = i;
    CHECK_GT(MAX_CUDA_DEVICES, shm->nprocesses);
  }
  CHECK(shm->nprocesses > 0) 
      << "No cuda device suppported";
  CHECK_EQ(shm->nprocesses, embedBuffers.size())
      << "Shared memory processes: " << shm->nprocesses 
      << ", send buffers: " << embedBuffers.size();

  _embedBuffers.assign(embedBuffers.begin(), embedBuffers.end());
  _denseBuffers.assign(denseBuffers.begin(), denseBuffers.end());

  _localEmbedBufLens.resize(localSize);
  _globalTotalEmbedBufLens.resize(workerNum, 0);

  // Allocate memory and an event for each process and fill 
  // the shared memory buffer with the IPC handles 
  for (size_t i = 0; i < shm->nprocesses; i++) {
    hipEvent_t event;
    CUDA_CALL(hipSetDevice(
        shm->devices[i]));
    CUDA_CALL(hipIpcGetMemHandle(
        (hipIpcMemHandle_t *)&shm->embedMemHandle[i], embedBuffers[i]));
    CUDA_CALL(hipIpcGetMemHandle(
        (hipIpcMemHandle_t *)&shm->denseMemHandle[i], denseBuffers[i]));

    CUDA_CALL(hipEventCreate(
        &event, hipEventDisableTiming | hipEventInterprocess));
    CUDA_CALL(hipIpcGetEventHandle(
        (hipIpcEventHandle_t *)&shm->eventHandle[i], event));
    
    // Store the buffers 
    _localEmbedBufLens[i] = embedBufferLens[i]; // local buffer length
  }
  _denseBufferLen = denseBufferLen;

  for (int i = 0; i < localSize; i++) {
    _globalTotalEmbedBufLens[workerID] += _localEmbedBufLens[i];
  }

  // Need a continous CPU buffer for each GPU
  _cpuBuffers.clear();
  for (int i = 0; i < localSize; i++) {
    void* _cpuBuffer;
    CUDA_CALL(hipHostAlloc(
        &_cpuBuffer, _denseBufferLen, hipHostMallocMapped | hipHostMallocPortable));
    _cpuBuffers.push_back(_cpuBuffer);
  }
  
  // The followings are for the global coordination of 
  // the embedding buffer length, which is equivalent to all-gather 
  auto ps = BytePSSparseCommon::GetPS();
  if (BytePSSparseCommon::IsDistributed()) {
    CHECK(ps); // must init the pslite instance before
    std::vector<ps::SArray<ps::Key>> tmpKeys;
    std::vector<ps::SArray<int>> tmpLens;
    std::vector<ps::SArray<char>> bufferLenSarrays;
    auto krs = ps::Postoffice::Get()->GetServerKeyRanges();
    for (int i = 0; i < workerNum; i++) {
      ps::Key key = i;
      int server = i;

      // vals
      ps::SArray<char> tmp(
          (char*)&_globalTotalEmbedBufLens[i], sizeof(size_t), false);
      bufferLenSarrays.push_back(tmp);
      
      // keys
      std::vector<ps::Key> tmp1(1, krs[server].begin() + key);
      ps::SArray<ps::Key> keys(tmp1);
      tmpKeys.push_back(keys);
      
      // lens
      std::vector<int> tmp2(1, sizeof(size_t));
      ps::SArray<int> lens(tmp2);
      tmpLens.push_back(lens);
    }

    // Push once to the associated server
    {
      int server = workerID;
      auto keys = tmpKeys[server];
      auto vals = bufferLenSarrays[server];
      auto lens = tmpLens[server];
      ps->Wait(ps->ZPush(keys, vals, lens));
    }

    // Call a barrier to sync across multiple workers. 
    // In case that some workers finish push too fast, 
    // and then pull from other workers too early
    ps::Postoffice::Get()->Barrier(
        0, ps::kWorkerGroup + ps::kServerGroup + ps::kScheduler);

    // Pull the embedding buffer length of other workers
    for (int key = 0; key < workerNum; key++) {
      int server = key;
      if (server == workerID) continue; // skip myself
      auto keys = tmpKeys[server];
      auto vals = bufferLenSarrays[server];
      auto lens = tmpLens[server];
      ps->Wait(ps->ZPull(keys, &vals, &lens));
    }
  }

  // For debug: print _localEmbedBufLens
  std::cout << "_localEmbedBufLens:" << std::endl;
  for (auto len : _localEmbedBufLens) 
    std::cout << len << " ";
  std::cout << std::endl;

  // For debug: print _globalTotalEmbedBufLens
  std::cout << "_globalTotalEmbedBufLens:" << std::endl;
  for (auto len : _globalTotalEmbedBufLens) 
    std::cout << len << " ";
  std::cout << std::endl;

  // Check the buffer size 
  size_t accmul = 0;
  for (auto len : _globalTotalEmbedBufLens) accmul += len / globalSize;
  CHECK_EQ(accmul, _denseBufferLen) << accmul << " " << _denseBufferLen;

  // Calc the global offset for the communication buffers
  size_t global_offset = 0;
  for (int id = 0; id < workerID; id++) {
    global_offset += _globalTotalEmbedBufLens[id] / globalSize;
  }

  // Prepare gossip-gather communication
  _local_gather_comms.resize(localSize);
  for (int i = 0; i < localSize; i++) {
    std::vector<float*> srcs(localSize);
    std::vector<size_t> srcs_lens(localSize);
    std::vector<size_t> send_counts(localSize);

    for (int j = 0; j < localSize; j++) {
      srcs[j] = 
          (float*)_embedBuffers[j] + 
          _localEmbedBufLens[j] / globalSize * (i + localSize * workerID);

      srcs_lens[j] = 
          _localEmbedBufLens[j] / globalSize * 
          (globalSize - (i + localSize * workerID));
          
      send_counts[j] = 
          _localEmbedBufLens[j] / globalSize;
    }
    float* dst = (float *)_denseBuffers[i] + global_offset;
    size_t dst_len = _globalTotalEmbedBufLens[workerID] / globalSize;

    std::string planfile_name("gather_plan_");
    planfile_name += std::to_string(i) + std::string(".json");
    _local_gather_comms[i] = std::make_unique<LocalGatherComm>(
        planfile_name, localSize, srcs, srcs_lens, send_counts, dst, dst_len);
  }

  // Prepare gossip-scatter communication
  _local_scatter_comms.resize(localSize);
  for (int i = 0; i < localSize; i++) {
    float* src = (float *)_denseBuffers[i] + global_offset;
    size_t src_len = _globalTotalEmbedBufLens[workerID] / globalSize;
    std::vector<float*> dsts(localSize);
    std::vector<size_t> dsts_lens(localSize);
    std::vector<size_t> send_counts(localSize);
    for (int j = 0; j < localSize; j++) {
      dsts[j] = 
          (float*)_embedBuffers[j] + 
          _localEmbedBufLens[j] / globalSize * (i + localSize * workerID);

      dsts_lens[j] = 
          _localEmbedBufLens[j] / globalSize * 
          (globalSize - (i + localSize * workerID));

      send_counts[j] = 
          _localEmbedBufLens[j] / globalSize;
    }

    std::string planfile_name("scatter_plan_");
    planfile_name += std::to_string(i) + std::string(".json");
    _local_scatter_comms[i] = std::make_unique<LocalScatterComm>(
        planfile_name, localSize, src, src_len, send_counts, dsts, dsts_lens);
  }
}

/*
void bytepsSparseInitDense(std::vector<void*>& denseDeltaBeforeReduceBuffers,
                           std::vector<void*>& denseDeltaAfterReduceBuffers,
                           int sizeDenseDelta) {

  CHECK_EQ(denseDeltaBeforeReduceBuffers.size(), denseDeltaAfterReduceBuffers.size());
  _denseDeltaBufferLength = sizeDenseDelta;

  // // Init IPC stuff
  // sharedMemoryInfo info;
  // CHECK_EQ(sharedMemoryCreate(bpsShmName, sizeof(shmStruct), &info), 0);
  // auto shm = (volatile shmStruct *)info.addr;
  // memset((void *)shm, 0, sizeof(*shm));

  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();

  // for (int i = 0; i < localSize; i++) {
  //   hipDeviceProp_t prop;
  //   CUDA_CALL(hipGetDeviceProperties(&prop, i));

  //   // CUDA IPC is only supported on devices with unified addressing
  //   if (!prop.unifiedAddressing) {
  //     // BPS_LOG(INFO) << "Device " << i << " does not support unified addressing, skipping...";
  //     continue;
  //   }
  //   // We require two processes accessing each device, so we need
  //   // to ensure exclusive or prohibited mode is not set
  //   if (prop.computeMode != hipComputeModeDefault) {
  //     // BPS_LOG(INFO) << "Device " << i << "is in an unsupported compute mode for this sample";
  //     continue;
  //   }

  //   shm->devices[shm->nprocesses++] = i;
  //   CHECK_GT(MAX_CUDA_DEVICES, shm->nprocesses);
  // }

  // CHECK(shm->nprocesses > 0) 
  //     << "No cuda device suppported";
  // CHECK_EQ(shm->nprocesses, embedBuffers.size())
  //     << "Shared memory processes: " << shm->nprocesses 
  //     << ", send buffers: " << embedBuffers.size();

  // We need to manually we need to clear the containers because
  // bytepsSparseInit() might be (unexpectedly) invoked multiple times
  _denseDeltaBeforeReduceBuffers.clear();
  _denseDeltaAfterReduceBuffers.clear();
  for (size_t i = 0; i < localSize; i++) {
    _denseDeltaBeforeReduceBuffers.push_back(denseDeltaBeforeReduceBuffers[i]); 
    _denseDeltaAfterReduceBuffers.push_back(denseDeltaAfterReduceBuffers[i]);
  }

  // Allocate memory and an event for each process and fill 
  // the shared memory buffer with the IPC handles 
  // for (size_t i = 0; i < shm->nprocesses; i++) {
  //   hipEvent_t event;
  //   CUDA_CALL(hipSetDevice(
  //       shm->devices[i]));

  //   CUDA_CALL(hipIpcGetMemHandle(
  //     (hipIpcMemHandle_t *)&shm->denseDeltaBeforeReduceMemHandle[i], denseDeltaBeforeReduceBuffers[i]));
  //   CUDA_CALL(hipIpcGetMemHandle(
  //     (hipIpcMemHandle_t *)&shm->denseDeltaAfterReduceMemHandle[i], denseDeltaAfterReduceBuffers[i]));

  //   // Store the buffers 
  //   _denseDeltaBeforeReduceBuffers.push_back(denseDeltaBeforeReduceBuffers[i]); 
  //   _denseDeltaAfterReduceBuffers.push_back(denseDeltaAfterReduceBuffers[i]);
  // }

  // Get CPU buffer for dense layer reduceasync
  {
    CUDA_CALL(hipHostAlloc(
        &_cpuDenseDeltaBuffers, sizeDenseDelta, hipHostMallocMapped | hipHostMallocPortable));
  }

  // Start the DenseReduce loop
  runDenseReduceLoop(_denseReduceLoop);
  _denseReducer = new ::byteps::common::CpuReducer(nullptr);
}
*/

extern "C" void bytepsSparseInitDensePerGPU(int device_id /* starts with 0 */,
                                            void* denseDeltaBeforeReduceBuffer,
                                            void* denseDeltaAfterReduceBuffer,
                                            int sizeDenseDelta) {
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  assert((device_id < localSize) && "Device id must be within local gpu size.");

  std::cout << "Init BytePS Sparse for dense layers: Device" << device_id << std::endl;

  if (device_id == 0){
    _denseDeltaBufferLength = sizeDenseDelta;
    _mtx_DenseLatestBuffers = new std::mutex();

    // Allocate latest parameter buffer.
    CUDA_CALL(hipHostAlloc(
      &_cpuDenseLatestBuffers, sizeDenseDelta, hipHostMallocMapped | hipHostMallocPortable));

    // Start the DenseReduce loop
    // _denseReducer = new ::byteps::common::CpuReducer(nullptr);
    // runDenseReduceLoop(_denseReduceLoop);

    // Start the 3-stage pipeline: D2H -> CpuReduce -> H2D
    _denseReducer = new ::byteps::common::CpuReducer(nullptr);
    runDenseReducePipeline(_denseD2HLoop, _denseReduceLoop, _denseH2DLoop, _denseReducer);
  } else{
    CHECK_EQ(_denseDeltaBufferLength, sizeDenseDelta);
  }

  // Get CPU buffer for dense layer reduceasync
  void * _cpuDenseDeltaBuffers_per_gpu;
  CUDA_CALL(hipHostAlloc(
    &_cpuDenseDeltaBuffers_per_gpu, sizeDenseDelta, hipHostMallocMapped | hipHostMallocPortable));
  _cpuDenseDeltaBuffers.push_back(_cpuDenseDeltaBuffers_per_gpu);
  _denseDeltaBeforeReduceBuffers.push_back(denseDeltaBeforeReduceBuffer); 
  _denseDeltaAfterReduceBuffers.push_back(denseDeltaAfterReduceBuffer);

  bool is_ready = false;
  std::mutex * mtx = new std::mutex();
  std::condition_variable * signal_cv = new std::condition_variable();
  _is_ready_per_gpu.push_back(is_ready);
  _signal_mtx_per_gpu.push_back(mtx);
  _signal_cv_per_gpu.push_back(signal_cv);
}


void bytepsSparseShutdown() {
}


void bytepsGatherExecAsync(int local_rank, hipStream_t stream) {
  // Gather from local peer GPUs on the same worker
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  auto workerNum = BytePSSparseCommon::GetNumWorker();

  _local_gather_comms[local_rank]->ExecAsync();
}

void bytepsSynchronize(int local_rank, hipStream_t stream, OP op) { 
  switch (op) {
    case GATHER:
      _local_gather_comms[local_rank]->Sync();
      break;
    case SCATTER:
      _local_scatter_comms[local_rank]->Sync();
      break;
    default:
      CHECK(0) << "unrecognized operation";
  }
  CUDA_CALL(hipStreamSynchronize(stream));
}

void bytepsScatterExecAsync(int local_rank, hipStream_t stream) {
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  void* baseSrcPtr = (void*)_denseBuffers[local_rank];

  _local_scatter_comms[local_rank]->ExecAsync();
}

// void dense_ready_callback(int local_rank) {
//   // std::mutex signal_mtx = _signal_mtx_per_gpu.at(local_rank);
//   // std::condition_variable signal_cv = _signal_cv_per_gpu.at(local_rank);

//   std::unique_lock<std::mutex> lck(* _signal_mtx_per_gpu.at(local_rank));
//   _is_ready_per_gpu.at(local_rank) = true;
//   _signal_cv_per_gpu.at(local_rank)->notify_one();
// }

// TODO (chengyu.dai): Add Broadcast for initializing the latestBuffer.
void bytepsDenseReduceExecAsync(int local_rank, hipStream_t stream) {
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  void* baseSrcPtr = (void*) (_denseDeltaBeforeReduceBuffers.at(local_rank));
  void* baseResultPtr = (void*) (_denseDeltaAfterReduceBuffers.at(local_rank));

  size_t buffer_size = _denseDeltaBufferLength;

  // Create a local thread and related mutex to synchronnize.
  _is_ready_per_gpu.at(local_rank) = false;

  // auto reduce_async_job = [//& signal_mtx, & signal_cv, & is_ready, 
  //                          local_rank, baseSrcPtr, baseResultPtr,
  //                          buffer_size, stream]() {
  //   // Copy dense layer's param delta D2H.
  //   CUDA_CALL(hipMemcpyAsync((void *)_cpuDenseDeltaBuffers, baseSrcPtr, buffer_size, hipMemcpyDeviceToHost, stream));
  //   CUDA_CALL(hipStreamSynchronize(stream));

  //   // CPU Work to reduce.
  //   _denseReducer->sum(_cpuDenseLatestBuffers, _cpuDenseDeltaBuffers, _denseDeltaBufferLength /* in bytes*/, DataType::BYTEPS_FLOAT32);

  //   // Copy dense layer's latest param H2D.
  //   CUDA_CALL(hipMemcpyAsync(baseResultPtr, _cpuDenseLatestBuffers, buffer_size, hipMemcpyHostToDevice, stream));
  //   CUDA_CALL(hipStreamSynchronize(stream));

  //   dense_ready_callback(local_rank);
  // };
  // _denseReduceLoop->add_worker(reduce_async_job);

  auto dense_ready_callback = 
    [] (int local_rank) {
    std::unique_lock<std::mutex> lck(* _signal_mtx_per_gpu.at(local_rank));
    _is_ready_per_gpu.at(local_rank) = true;
    _signal_cv_per_gpu.at(local_rank)->notify_one();
  };

  DenseTask task;
  {
    task.workerID = workerID;
    task.local_rank = local_rank;
    task.buffer_size = buffer_size; // In bytes.
    task.streamH2D = stream;
    task.streamD2H = stream; // TODO(chengyu.dai): separate the streams for two directions.

    task.baseSrcPtr = baseSrcPtr;
    task.cpuDenseDeltaPtr = (void *) (_cpuDenseDeltaBuffers.at(local_rank));
    task.cpuDenseLatestPtr = _cpuDenseLatestBuffers;
    task.baseResultPtr = baseResultPtr;

    task.allFinishCallback = dense_ready_callback;
  }
  _denseD2HLoop->add_predefined_worker(task);
}

void bytepsDenseSynchronize(int local_rank, hipStream_t stream) {
  // auto signal_mtx = _signal_mtx_per_gpu.at(local_rank);
  // std::condition_variable signal_cv = _signal_cv_per_gpu.at(local_rank);

  std::unique_lock<std::mutex> lck(* _signal_mtx_per_gpu.at(local_rank));
  while (!_is_ready_per_gpu.at(local_rank))
    _signal_cv_per_gpu.at(local_rank)->wait(lck);
}


} // namespace sparse
} // namespace byteps 