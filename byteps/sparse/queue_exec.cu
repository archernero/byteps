#include "queue_exec.h"

namespace byteps {
namespace sparse {

//********** QueueExecLoop **********//

void QueueExecLoop::start_executors() {
  // The initialization will be ignored if the poller is already up.
  if (background_job_polls_.size() >= 1)
    return;

  running_ = true;

  auto background_job = [this] () {
    while (running_) {
      std::function<void()> job = nullptr;
      // The job is not executed in the critical section, otherwise it is not
      // different from mutex lock the whole thing.
      {
        std::unique_lock<std::mutex> lck(job_queue_mtx_);
        while (job_queue_.empty())
          job_queue_cv_.wait(lck);

        job = job_queue_.front();
        job_queue_.pop();
      }
      if (job != nullptr) {
        job();
      }
    }
  };
  background_job_polls_.emplace_back(background_job);
}

void QueueExecLoop::stop_executors() {
  running_ = false;
  auto exit_job = []() { 
      // BPS_LOG(INFO) << "Exiting the executor."; 
  };
  add_worker(exit_job);

  if (!background_job_polls_.empty()) {
    for (auto& job : background_job_polls_) {
      job.join();
    }
    background_job_polls_.clear();
  }
}

void QueueExecLoop::add_worker(std::function<void()> job) {
  std::unique_lock<std::mutex> lck(job_queue_mtx_);
  job_queue_.push(std::move(job));
  job_queue_cv_.notify_all();
}

QueueExecLoop* QueueExecLoop::init_loop() {
  auto queue_exec_ptr = new QueueExecLoop();
  queue_exec_ptr->start_executors();
  return queue_exec_ptr;
}


//********** PredefinedQueueExecLoop **********//


void PredefinedQueueExecLoop::add_predefined_worker(DenseTask task){
  std::function<void()> job = [this, task] () {
    predefined_work(task);
    if (downstream_ != nullptr)
      downstream_->add_predefined_worker(task);
    else
      task.allFinishCallback(task.local_rank);
  };
  add_worker(job);
}

void PredefinedQueueExecLoop::set_downstream(PredefinedQueueExecLoop * downstream) {
  downstream_ = downstream;
}


//********** MemcpyH2DQueueExecLoop **********//

void MemcpyH2DQueueExecLoop::predefined_work(DenseTask task) {
  // Copy dense layer's param delta D2H.
  CUDA_CALL(hipMemcpyAsync(task.cpuDenseDeltaPtr, task.baseSrcPtr,
                            task.buffer_size, hipMemcpyDeviceToHost, task.streamD2H));
  CUDA_CALL(hipStreamSynchronize(task.streamD2H));
}

MemcpyH2DQueueExecLoop * MemcpyH2DQueueExecLoop::init_loop(){
  auto loop = new MemcpyH2DQueueExecLoop();
  loop->start_executors();
  return loop;
}

//********** CPUReduceQueueExecLoop **********//

void CPUReduceQueueExecLoop::predefined_work(DenseTask task) {
  // CPU Work to reduce.
  _loopdenseReducer->sum(task.cpuDenseLatestPtr, task.cpuDenseDeltaPtr, task.buffer_size /* in bytes*/, DataType::BYTEPS_FLOAT32);
}

CPUReduceQueueExecLoop * CPUReduceQueueExecLoop::init_loop(::byteps::common::CpuReducer* denseReducer){
  auto loop = new CPUReduceQueueExecLoop(denseReducer);
  loop->start_executors();
  return loop;
}

//********** MemcpyD2HQueueExecLoop **********//

void MemcpyD2HQueueExecLoop::predefined_work(DenseTask task) {
  // Copy dense layer's latest param H2D.
  CUDA_CALL(hipMemcpyAsync(task.baseResultPtr, task.cpuDenseLatestPtr, 
                            task.buffer_size, hipMemcpyHostToDevice, task.streamD2H));
  CUDA_CALL(hipStreamSynchronize(task.streamD2H));
}

MemcpyD2HQueueExecLoop * MemcpyD2HQueueExecLoop::init_loop(){
  auto loop = new MemcpyD2HQueueExecLoop();
  loop->start_executors();
  return loop;
}

} // namespace sparse
} // namespace byteps 
